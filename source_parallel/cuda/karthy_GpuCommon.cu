#include "karthy_GpuCommon.h"

void karthy::cuda::printError(void)
{
#if DEBUG
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}
#endif
}