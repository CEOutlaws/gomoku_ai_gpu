#include "hip/hip_runtime.h"
#include "karthy_AiGpu.h"

karthy::cuda::AiGpu::AiGpu()
{
	hipMalloc(&this->d_getSymmetricTypeResult, sizeof(int));  PRINT_ERROR();
}

__global__ void getSymmetricTypeKernel(
	int* result,
	karthy::cuda::GpuMatrix matrix1, 
	karthy::cuda::GpuMatrix matrix2,
	karthy::cuda::GpuAction actionForCheck,
	karthy::cuda::GpuActionArray actionArray)
{
	//int row = threadIdx.y; 
	//int col = threadIdx.x;
	__shared__ int s_mat1[BOARD_SIDE_LENGTH][BOARD_SIDE_LENGTH];
	__shared__ int s_mat2[BOARD_SIDE_LENGTH][BOARD_SIDE_LENGTH];
	__shared__ int s_equalMatrixResult;
	__shared__ int s_equalPointResult;
	__shared__ karthy::cuda::GpuAction s_arrayAction;

	s_mat1[threadIdx.y][threadIdx.x] = matrix1.d_element[threadIdx.y * blockDim.x + threadIdx.x];
	s_mat2[threadIdx.y][threadIdx.x] = matrix2.d_element[threadIdx.y * blockDim.x + threadIdx.x];
	/*
#if DEBUG
	if (blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.y == 0)
	{
		printf("smat1\n");

		for (int boxIndexY = 0; boxIndexY < BOARD_SIDE_LENGTH; boxIndexY++)
		{
			for (int boxIndexX = 0; boxIndexX < BOARD_SIDE_LENGTH; boxIndexX++)
			{
				printf("%d", s_mat1[boxIndexY][boxIndexX]);
			}
			printf("\n");
		}
		printf("smat2\n");
		for (int boxIndexY = 0; boxIndexY < BOARD_SIDE_LENGTH; boxIndexY++)
		{
			for (int boxIndexX = 0; boxIndexX < BOARD_SIDE_LENGTH; boxIndexX++)
			{
				printf("%d", s_mat2[boxIndexY][boxIndexX]);
			}
			printf("\n");
		}
	}
#endif
*/

	//mat2 vs mat1 ROTATE_000_FLIP_0
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		s_arrayAction.x = actionArray.d_elementX[blockIdx.x];
		s_arrayAction.y = actionArray.d_elementY[blockIdx.x];

		s_mat1[s_arrayAction.y][s_arrayAction.x] = actionArray.taker;
		s_mat2[actionForCheck.y][actionForCheck.x] = actionArray.taker;

		s_equalMatrixResult = 1;
		s_equalPointResult =
			s_arrayAction.x == actionForCheck.x &&
			s_arrayAction.y == actionForCheck.y;
	}

	__syncthreads();

	if (s_equalPointResult)
	{
		if (s_mat1[threadIdx.y][threadIdx.x] != s_mat2[threadIdx.y][threadIdx.x])
		{
			s_equalMatrixResult = 0;
		}

		__syncthreads();

		if (s_equalMatrixResult)
		{
			if (threadIdx.x == 0 && threadIdx.y == 0)
			{
				*result = 1;
			}
			return;
		}
	}

	//mat2 vs mat1 ROTATE_090_FLIP_0
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		s_equalMatrixResult = 1;
		s_equalPointResult =
			s_arrayAction.x == actionForCheck.y &&
			s_arrayAction.y == -actionForCheck.x + BOARD_SIDE_LENGTH - 1;
	}

	__syncthreads();

	if (s_equalPointResult)
	{
		if (s_mat1[threadIdx.y][threadIdx.x] != s_mat2[threadIdx.x][BOARD_SIDE_LENGTH - 1 - threadIdx.y])
		{
			s_equalMatrixResult = 0;
		}
		__syncthreads();

		if (s_equalMatrixResult)
		{
			if (threadIdx.x == 0 && threadIdx.y == 0)
			{
				*result = 2;
			}
			return;
		}
	}

	//mat2 vs mat1 ROTATE_180_FLIP_0
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		s_equalMatrixResult = 1;
		s_equalPointResult =
			s_arrayAction.x == BOARD_SIDE_LENGTH - 1 - actionForCheck.x &&
			s_arrayAction.y == BOARD_SIDE_LENGTH - 1 - actionForCheck.y;
	}

	__syncthreads();

	if (s_equalPointResult)
	{
		if (s_mat1[threadIdx.y][threadIdx.x] != s_mat2[BOARD_SIDE_LENGTH - 1 - threadIdx.y][BOARD_SIDE_LENGTH - 1 - threadIdx.x])
		{
			s_equalMatrixResult = 0;
		}
		__syncthreads();

		if (s_equalMatrixResult)
		{
			if (threadIdx.x == 0 && threadIdx.y == 0)
			{
				*result = 3;
			}
			return;
		}
	}

	//mat2 vs mat1 ROTATE_270_FLIP_0
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		s_equalMatrixResult = 1;
		s_equalPointResult =
			s_arrayAction.x == BOARD_SIDE_LENGTH - 1 - actionForCheck.y &&
			s_arrayAction.y == actionForCheck.x;
	}

	__syncthreads();

	if (s_equalPointResult)
	{
		if (s_mat1[threadIdx.y][threadIdx.x] != s_mat2[-threadIdx.x + BOARD_SIDE_LENGTH - 1][threadIdx.y])
		{
			s_equalMatrixResult = 0;
		}
		__syncthreads();

		if (s_equalMatrixResult)
		{
			if (threadIdx.x == 0 && threadIdx.y == 0)
			{
				*result = 4;
			}
			return;
		}
	}

	//mat2 vs mat1 0 ROTATE_000_FLIP_1

	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		s_equalMatrixResult = 1;
		s_equalPointResult =
			s_arrayAction.x == -actionForCheck.x + BOARD_SIDE_LENGTH - 1 &&
			s_arrayAction.y == actionForCheck.y;
	}

	__syncthreads();

	if (s_equalPointResult)
	{
		if (s_mat1[threadIdx.y][threadIdx.x] != s_mat2[threadIdx.y][-threadIdx.x + BOARD_SIDE_LENGTH - 1])
		{
			s_equalMatrixResult = 0;
		}
		__syncthreads();

		if (s_equalMatrixResult)
		{
			if (threadIdx.x == 0 && threadIdx.y == 0)
			{
				*result = 5;
			}
			return;
		}
	}
	//mat2 vs mat1 ROTATE_090_FLIP_1
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		s_equalMatrixResult = 1;
		s_equalPointResult =
			s_arrayAction.x == actionForCheck.y &&
			s_arrayAction.y == actionForCheck.x;
	}

	__syncthreads();

	if (s_equalPointResult)
	{
		if (s_mat1[threadIdx.y][threadIdx.x] != s_mat2[threadIdx.x][threadIdx.y])
		{
			s_equalMatrixResult = 0;
		}
		__syncthreads();

		if (s_equalMatrixResult)
		{
			if (threadIdx.x == 0 && threadIdx.y == 0)
			{
				*result = 6;
			}
			return;
		}
	}

	//mat2 vs mat1 ROTATE_180_FLIP_1
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		s_equalMatrixResult = 1;
		s_equalPointResult =
			s_arrayAction.x == actionForCheck.x &&
			s_arrayAction.y == -actionForCheck.y + BOARD_SIDE_LENGTH - 1;
	}

	__syncthreads();

	if (s_equalPointResult)
	{
		if (s_mat1[threadIdx.y][threadIdx.x] != s_mat2[-threadIdx.y + BOARD_SIDE_LENGTH - 1][threadIdx.x])
		{
			s_equalMatrixResult = 0;
		}
		__syncthreads();

		if (s_equalMatrixResult)
		{
			if (threadIdx.x == 0 && threadIdx.y == 0)
			{
				*result = 7;
			}
			return;
		}
	}

	//mat2 vs mat1 ROTATE_270_FLIP_1
	if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		s_equalMatrixResult = 1;
		s_equalPointResult = 
			s_arrayAction.x == -actionForCheck.y + BOARD_SIDE_LENGTH - 1 &&
			s_arrayAction.y == -actionForCheck.x + BOARD_SIDE_LENGTH - 1;
	}

	__syncthreads();

	if (s_equalPointResult)
	{
		if (s_mat1[threadIdx.y][threadIdx.x] != s_mat2[-threadIdx.x + BOARD_SIDE_LENGTH - 1][-threadIdx.y + BOARD_SIDE_LENGTH - 1])
		{
			s_equalMatrixResult = 0;
		}
		__syncthreads();

		if (s_equalMatrixResult)
		{
			if (threadIdx.x == 0 && threadIdx.y == 0)
			{
				*result = 8;
			}
			return;
		}
	}
}

int karthy::cuda::AiGpu::getSymmetricType(const GpuAction& action, const GpuMatrix& onActionMatrix)
{
	if (this->actionArray.getElementCount() == 0)
	{
		return 0;
	}

	int result;

	dim3 dimBlock(BOARD_SIDE_LENGTH, BOARD_SIDE_LENGTH);
	dim3 dimGrid(this->actionArray.getElementCount());

	hipMemset(this->d_getSymmetricTypeResult, 0, sizeof(int)); PRINT_ERROR();

	getSymmetricTypeKernel << <dimGrid, dimBlock >> > (this->d_getSymmetricTypeResult, this->logicalMatrix, onActionMatrix,
		action, this->actionArray); PRINT_ERROR();

	hipMemcpy(&result, d_getSymmetricTypeResult, sizeof(int), hipMemcpyDeviceToHost); PRINT_ERROR();

	return result;
}

void karthy::cuda::AiGpu::reset(void)
{
	this->actionArray.clearAll();
	this->logicalMatrix.setAllElement(2);
	this->physicalMatrix.setAllElement(2);
}
