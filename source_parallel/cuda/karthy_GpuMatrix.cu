#include "hip/hip_runtime.h"
#include "karthy_GpuMatrix.h"

karthy::cuda::GpuMatrix::GpuMatrix()
{
	hipMalloc(&this->d_element, BOARD_SIDE_LENGTH * BOARD_SIDE_LENGTH * sizeof(int)); PRINT_ERROR();
}

void karthy::cuda::GpuMatrix::setElement(const karthy::cuda::GpuAction& action, int taker)
{
	setElement(action.y, action.x, taker);
}

void karthy::cuda::GpuMatrix::setElement(int row, int col, int value)
{
	hipMemcpy(&this->d_element[row * BOARD_SIDE_LENGTH + col], &value, sizeof(int), hipMemcpyHostToDevice); PRINT_ERROR();
}

__global__ void setAllElementKernel(int* d_element, int value)
{
	d_element[blockIdx.x * blockDim.x + threadIdx.x] = value;
}

void karthy::cuda::GpuMatrix::setAllElement(int value)
{
	dim3 dimBlock(BOARD_SIDE_LENGTH);
	dim3 dimGrid(BOARD_SIDE_LENGTH);

	setAllElementKernel << <dimGrid, dimBlock >> > (this->d_element, value); PRINT_ERROR();
}

void karthy::cuda::GpuMatrix::print(void)
{
#if DEBUG
	for (int boxIndexY = 0; boxIndexY < BOARD_SIDE_LENGTH; boxIndexY++)
	{
		for (int boxIndexX = 0; boxIndexX < BOARD_SIDE_LENGTH; boxIndexX++)
		{
			int value;
			hipMemcpy(&value, &this->d_element[boxIndexY * BOARD_SIDE_LENGTH + boxIndexX], sizeof(int), hipMemcpyDeviceToHost); PRINT_ERROR();
			printf("%d", value);
		}
		printf("\n");
	}
#endif
}
