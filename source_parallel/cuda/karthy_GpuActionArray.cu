#include "hip/hip_runtime.h"
#include "karthy_GpuActionArray.h"

karthy::cuda::GpuActionArray::GpuActionArray()
{
	hipMalloc(&this->d_searchActionSymmetricResult, sizeof(karthy::cuda::GpuAction)); PRINT_ERROR();

	size_t size = BOARD_SIDE_LENGTH * BOARD_SIDE_LENGTH * sizeof(int);
	hipMalloc(&this->d_elementX, size); PRINT_ERROR();
	hipMalloc(&this->d_elementY, size); PRINT_ERROR();

	this->elementQValue = new double[BOARD_SIDE_LENGTH * BOARD_SIDE_LENGTH];
	this->nextStateId = new uint64_t[BOARD_SIDE_LENGTH * BOARD_SIDE_LENGTH];
}


void karthy::cuda::GpuActionArray::addAction(const karthy::cuda::GpuAction& actionForAdd)
{
	//hipMemset(&this->d_elementX[this->elementCount], actionForAdd.x, sizeof(int)); PRINT_ERROR();
	//hipMemset(&this->d_elementY[this->elementCount], actionForAdd.y, sizeof(int)); PRINT_ERROR();

	hipMemcpy(&this->d_elementX[this->elementCount], &actionForAdd.x, sizeof(int), hipMemcpyHostToDevice); PRINT_ERROR();
	hipMemcpy(&this->d_elementY[this->elementCount], &actionForAdd.y, sizeof(int), hipMemcpyHostToDevice); PRINT_ERROR();

	this->elementQValue[this->elementCount] = actionForAdd.qValue;
	this->nextStateId[this->elementCount] = actionForAdd.nextStateId;

	if (this->bestAction.isNull() || this->bestAction.qValue < actionForAdd.qValue)
	{
		this->bestAction = actionForAdd;
		this->bestAction.order = elementCount;
	}

	if (this->exploreAction.isNull() && actionForAdd.qValue == 0)
	{
		this->exploreAction = actionForAdd;
		this->exploreAction.order = elementCount;
	}

	this->elementCount++;
}

void karthy::cuda::GpuActionArray::clearAll(void)
{
	this->elementCount = 0;
	this->bestAction.setNull();
	this->exploreAction.setNull();
}
void karthy::cuda::GpuActionArray::print(void)
{
#if DEBUG
	int x;
	int y;

	for (int index = 0; index < this->elementCount; index++)
	{
		hipMemcpy(&x, &this->d_elementX[index], sizeof(int), hipMemcpyDeviceToHost); PRINT_ERROR();
		hipMemcpy(&y, &this->d_elementY[index], sizeof(int), hipMemcpyDeviceToHost); PRINT_ERROR();
		printf("(%d, %d)\n", x, y);
	}
#endif
}
template <int symmetricType>
__global__ void searchActionSymmetricKernel(
	karthy::cuda::GpuAction* d_searchActionSymmetricResult,
	karthy::cuda::GpuActionArray actionArray,
	karthy::cuda::GpuAction actionForSearch
	)
{
	const int actionIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (actionIdx >= actionArray.getElementCount()) return;

	//ROTATE 0 FLIP 0
	if (symmetricType == 1)
	{
		if (actionArray.d_elementX[actionIdx] == actionForSearch.x &&
			actionArray.d_elementY[actionIdx] == actionForSearch.y)
		{
			d_searchActionSymmetricResult->x = actionForSearch.x;
			d_searchActionSymmetricResult->y = actionForSearch.y;
			d_searchActionSymmetricResult->order = actionIdx;
			return;
		}
	}

	//ROTATE 90 FLIP 0
	if (symmetricType == 2)
	{
		if (actionArray.d_elementX[actionIdx] == actionForSearch.y &&
			actionArray.d_elementY[actionIdx] == -actionForSearch.x + BOARD_SIDE_LENGTH - 1)
		{
			d_searchActionSymmetricResult->x = actionForSearch.y;
			d_searchActionSymmetricResult->y = -actionForSearch.x + BOARD_SIDE_LENGTH - 1;
			d_searchActionSymmetricResult->order = actionIdx;
			return;
		}
	}

	//ROTATE 180 FLIP 0
	if (symmetricType == 3)
	{
		if (actionArray.d_elementX[actionIdx] == BOARD_SIDE_LENGTH - 1 - actionForSearch.x &&
			actionArray.d_elementY[actionIdx] == BOARD_SIDE_LENGTH - 1 - actionForSearch.y)
		{
			d_searchActionSymmetricResult->x = BOARD_SIDE_LENGTH - 1 - actionForSearch.x;
			d_searchActionSymmetricResult->y = BOARD_SIDE_LENGTH - 1 - actionForSearch.y;
			d_searchActionSymmetricResult->order = actionIdx;
			return;
		}
	}

	//ROTATE 270 FLIP 0
	if (symmetricType == 4)
	{
		if (actionArray.d_elementX[actionIdx] == BOARD_SIDE_LENGTH - 1 - actionForSearch.y &&
			actionArray.d_elementY[actionIdx] == actionForSearch.x)
		{
			d_searchActionSymmetricResult->x = BOARD_SIDE_LENGTH - 1 - actionForSearch.y;
			d_searchActionSymmetricResult->y = actionForSearch.x;
			d_searchActionSymmetricResult->order = actionIdx;
			return;
		}
	}

	//ROTATE 0 FLIP 1
	if (symmetricType == 5)
	{
		if (actionArray.d_elementX[actionIdx] == -actionForSearch.x + BOARD_SIDE_LENGTH - 1 &&
			actionArray.d_elementY[actionIdx] == actionForSearch.y)
		{
			d_searchActionSymmetricResult->x = -actionForSearch.x + BOARD_SIDE_LENGTH - 1;
			d_searchActionSymmetricResult->y = actionForSearch.y;
			d_searchActionSymmetricResult->order = actionIdx;
			return;
		}
	}

	//ROTATE 90 FLIP 1
	if (symmetricType == 6)
	{
		if (actionArray.d_elementX[actionIdx] == actionForSearch.y &&
			actionArray.d_elementY[actionIdx] == actionForSearch.x)
		{
			d_searchActionSymmetricResult->x = actionForSearch.y;
			d_searchActionSymmetricResult->y = actionForSearch.x;
			d_searchActionSymmetricResult->order = actionIdx;
			return;
		}
	}

	//ROTATE 180 FLIP 1
	if (symmetricType == 7)
	{
		if (actionArray.d_elementX[actionIdx] == actionForSearch.x &&
			actionArray.d_elementY[actionIdx] == -actionForSearch.y + BOARD_SIDE_LENGTH - 1)
		{
			d_searchActionSymmetricResult->x = actionForSearch.x;
			d_searchActionSymmetricResult->y = -actionForSearch.y + BOARD_SIDE_LENGTH - 1;
			d_searchActionSymmetricResult->order = actionIdx;
			return;
		}
	}

	//ROTATE 270 FLIP 1
	if (symmetricType == 8)
	{
		if (actionArray.d_elementX[actionIdx] == -actionForSearch.y + BOARD_SIDE_LENGTH - 1 &&
			actionArray.d_elementY[actionIdx] == -actionForSearch.x + BOARD_SIDE_LENGTH - 1)
		{
			d_searchActionSymmetricResult->x = -actionForSearch.y + BOARD_SIDE_LENGTH - 1;
			d_searchActionSymmetricResult->y = -actionForSearch.x + BOARD_SIDE_LENGTH - 1;
			d_searchActionSymmetricResult->order = actionIdx;
			return;
		}
	}
}

karthy::cuda::GpuAction karthy::cuda::GpuActionArray::searchActionSymmetric(const karthy::cuda::GpuAction& actionForSearch, int symmetricType)
{
	karthy::cuda::GpuAction result;

	dim3 dimBlock(BOARD_SIDE_LENGTH);
	dim3 dimGrid(BOARD_SIDE_LENGTH);

	hipMemset(this->d_searchActionSymmetricResult, 0, sizeof(int));

	switch (symmetricType)
	{
	case(1):
		searchActionSymmetricKernel<1> << <dimGrid, dimBlock >> > (this->d_searchActionSymmetricResult, (*this), actionForSearch);
		break;
	case(2):
		searchActionSymmetricKernel<2> << <dimGrid, dimBlock >> > (this->d_searchActionSymmetricResult, (*this), actionForSearch);
		break;
	case(3):
		searchActionSymmetricKernel<3> << <dimGrid, dimBlock >> > (this->d_searchActionSymmetricResult, (*this), actionForSearch);
		break;
	case(4):
		searchActionSymmetricKernel<4> << <dimGrid, dimBlock >> > (this->d_searchActionSymmetricResult, (*this), actionForSearch);
		break;
	case(5):
		searchActionSymmetricKernel<5> << <dimGrid, dimBlock >> > (this->d_searchActionSymmetricResult, (*this), actionForSearch);
		break;
	case(6):
		searchActionSymmetricKernel<6> << <dimGrid, dimBlock >> > (this->d_searchActionSymmetricResult, (*this), actionForSearch);
		break;
	case(7):
		searchActionSymmetricKernel<7> << <dimGrid, dimBlock >> > (this->d_searchActionSymmetricResult, (*this), actionForSearch);
		break;
	case(8):
		searchActionSymmetricKernel<8> << <dimGrid, dimBlock >> > (this->d_searchActionSymmetricResult, (*this), actionForSearch);
		break;
	}

	hipMemcpy(&result, this->d_searchActionSymmetricResult, sizeof(karthy::cuda::GpuAction), hipMemcpyDeviceToHost);
	result.nextStateId = this->nextStateId[result.order];

	return result;
}

karthy::cuda::GpuAction karthy::cuda::GpuActionArray::getBestAction(void)
{
	return this->bestAction;
}

karthy::cuda::GpuAction karthy::cuda::GpuActionArray::getExploreAction(void)
{
	return this->exploreAction;
}

karthy::cuda::GpuAction karthy::cuda::GpuActionArray::getActionByIndex(int index)
{
	karthy::cuda::GpuAction result;

	hipMemcpy(&result.x, &this->d_elementX[index], sizeof(int), hipMemcpyDeviceToHost); PRINT_ERROR();
	hipMemcpy(&result.y, &this->d_elementY[index], sizeof(int), hipMemcpyDeviceToHost); PRINT_ERROR();

	//printf("%d %d\n", index, this->elementCount);
	//printf("%f\n", this->elementQValue[index]);

	result.qValue = this->elementQValue[index];
	result.nextStateId = this->nextStateId[index];
	result.order = index;

	return result;
}

int karthy::cuda::GpuActionArray::getElementCount(void)
{
	return elementCount;
}
